#include "hip/hip_runtime.h"
#ifdef XCPU
#define __device__
#define __global__

#define rsqrtf 1/sqrtf
#define __float2int_rn (int)lroundf
#define __float2int_ru (int)ceilf
#define __float2int_rd (int)floorf

struct int2{
  int x, y;
};

struct uint4{
  unsigned int x, y, z, w;
};

float int_as_float(unsigned int x){
  union{
    unsigned int i;
    float f;
  };
  i=x; return f;
}

unsigned int atomicAdd(unsigned int * i, unsigned int j){
  unsigned int k=*i; *i+=j;
  return k;
}

struct ThreadIdx{
  int x;
} threadIdx;

struct BlockDim{
  int x;
} blockDim;

unsigned int seed=0;

#if defined(__APPLE_CC__) || defined(__FreeBSD__)
void sincosf(float x, float * s, float * c){ *s = sin(x); *c = cos(x); }
#endif
#endif

__device__ float xrnd(uint4 & s){
  unsigned int tmp;
  do{
    unsigned long long sda;
#ifdef XCPU
    sda = s.z * (unsigned long long) s.x;
#else
    asm("mul.wide.u32 %0, %1, %2;" : "=l"(sda) : "r"(s.x), "r"(s.z));
#endif
    sda += s.y; s.x = sda; s.y = sda >> 32; tmp = s.x >> 9;
  } while(tmp==0);
  return int_as_float(tmp|0x3f800000)-1.0f;
}

#ifdef LONG
__device__ float mrnd(float k, uint4 & s){  // gamma distribution
  float x;
  if(k<1){  // Weibull algorithm
    float c=1/k;
    float d=(1-k)*powf(k, k/(1-k));
    float z, e;
    do{
      z=-logf(xrnd(s));
      e=-logf(xrnd(s));
      x=powf(z, c);
    } while(z+e<d+x);
  }
  else{  // Cheng's algorithm
    float b=k-logf(4.0f);
    float l=sqrtf(2*k-1);
    float c=1+logf(4.5f);
    float u, v, y, z, r;
    do{
      u=xrnd(s); v=xrnd(s);
      y=logf(v/(1-v))/l;
      x=k*expf(y);
      z=u*v*v;
      r=b+(k+l)*y-x;
    } while(r<4.5f*z-c && r<logf(z));
  }
  return x;
}
#endif

__device__ void swap(float & x, float & y){
  float a=x; x=y; y=a;
}

__device__ void rotate(float & cs, float & si, float3 & n, uint4 & s){
  float3 p1, p2;
  int i=0;
  {
    float3 r;
    r.x=n.x*n.x, r.y=n.y*n.y, r.z=n.z*n.z;
    if(r.y>r.z){
      if(r.y>r.x) i=(swap(n.x,n.y),swap(r.x,r.y),1);
    }
    else{
      if(r.z>r.x) i=(swap(n.x,n.z),swap(r.x,r.z),2);
    }

    r.y=rsqrtf(r.x+r.y); p1.x=-n.y*r.y; p1.y=n.x*r.y; p1.z=0;
    r.z=rsqrtf(r.x+r.z); p2.x=-n.z*r.z; p2.y=0; p2.z=n.x*r.z;
  }

  {
    float4 q1;

    q1.x=p1.x-p2.x; q1.y=p1.y-p2.y; q1.z=p1.z-p2.z;
    p2.x+=p1.x; p2.y+=p1.y; p2.z+=p1.z;

    q1.w=rsqrtf(q1.x*q1.x+q1.y*q1.y+q1.z*q1.z);
    p1.x=q1.x*q1.w; p1.y=q1.y*q1.w; p1.z=q1.z*q1.w;

    q1.w=rsqrtf(p2.x*p2.x+p2.y*p2.y+p2.z*p2.z);
    p2.x*=q1.w; p2.y*=q1.w; p2.z*=q1.w;
  }

  {
    float2 p;
    float xi=2*FPI*xrnd(s);
    sincosf(xi, &p.y, &p.x);

    n.x=cs*n.x+si*(p.x*p1.x+p.y*p2.x);
    n.y=cs*n.y+si*(p.x*p1.y+p.y*p2.y);
    n.z=cs*n.z+si*(p.x*p1.z+p.y*p2.z);

    float r=rsqrtf(n.x*n.x+n.y*n.y+n.z*n.z);
    n.x*=r; n.y*=r; n.z*=r;
    if(i==1) swap(n.x,n.y); else if(i==2) swap(n.x,n.z);
  }
}

#ifdef TILT
#ifndef XCPU
__device__ int __float2int_rd(float x);
__host__ int __float2int_rd(float x){ return (int)floorf(x); }
__host__
#endif

__device__ float zshift(dats & d, float4 & r){
  if(d.lnum==0) return 0;
  float z=(r.z-d.lmin)*d.lrdz;
  int k=min(max(__float2int_rd(z), 0), d.lpts-2);
  int l=k+1;

  float nr=d.lnx*r.x+d.lny*r.y-d.r0;
  for(int j=1; j<LMAX; j++) if(nr<d.lr[j] || j==d.lnum-1){
    int i=j-1;
    return ( (d.lp[j][l]*(z-k)+d.lp[j][k]*(l-z))*(nr-d.lr[i]) +
	     (d.lp[i][l]*(z-k)+d.lp[i][k]*(l-z))*(d.lr[j]-nr) )/(d.lr[j]-d.lr[i]);
  }
  return 0;
}
#endif

__device__ void ctr(dats & d, float2 & r, float2 & p){
#ifdef ROMB
  p.x=d.cb[0][0]*r.x+d.cb[1][0]*r.y;
  p.y=d.cb[0][1]*r.x+d.cb[1][1]*r.y;
#else
  p=r;
#endif
}

#ifdef XCPU
DOM * oms;
#else
__constant__ DOM oms[MAXGEO];

__device__ inline unsigned int smid(){
  unsigned int r;
  asm("mov.u32 %0, %smid;" : "=r"(r));
  return r;
}
#endif

#if defined(USMA) && defined(RAND)
#define XINC i=atomicAdd(&eidx, e.gridDim)
#define XIDX e.gridDim*blockDim.x+e.blockIdx
#else
#define XINC i+=eidx
#define XIDX e.gridDim*blockDim.x
#endif

#ifdef HOLE
#define IFH(x,y) x
#else
#define IFH(x,y) y
#endif

__global__ void propagate(dats * ed, unsigned int num){
  uint4 s;
  unsigned int niw=0;
#ifdef XCPU
  float3 n;
  float4 r;
  dats & e = * ed;
  static unsigned int eidx;
  if(threadIdx.x==0) eidx = XIDX;
#else
  float3 n={0,0,0};
  float4 r={0,0,0,0};
  __shared__ dats e;
  unsigned int & eidx = e.hidx;

  if(num==0){
    ed->hidx=0;
    ed->tn=-1U;
    ed->tx=0;
    ed->ab=0;
    ed->mp=0;
    __threadfence();
    return;
  }

  if(threadIdx.x==0){
    e=*ed; e.tn=clock();
    e.blockIdx=smid()==e.blockIdx?-1:(int)atomicAdd(&ed->mp, 1);
    eidx=XIDX;
  }
  __syncthreads();

  if(e.blockIdx==-1) return;
#endif

  ices * w;
  const unsigned int idx=threadIdx.x*e.gridDim+e.blockIdx;

  {
#ifndef XCPU
    const unsigned int & seed = idx;
#endif
    s.w=seed%e.rsize;
    s.x=e.z->rs[s.w];
    s.y=e.z->rs[s.w] >> 32;
    s.z=e.z->rm[s.w];
  }

  int old;
  float TOT=0, IFH(SCA,sca);

#ifdef TALL
  for(unsigned int i=idx; i<num; i+=e.gridDim*blockDim.x){
#else
  for(unsigned int i=idx; i<num; TOT==0 && (XINC)){
    int om=-1;
    if(TOT==0){ // initialize photon
#endif

      unsigned int j=min(__float2int_rd(WNUM*xrnd(s)), WNUM-1);
      w=&e.z->w[j];
      if(e.type>0){
	r.x=e.r[0];
	r.y=e.r[1];
	r.z=e.r[2];
	r.w=0;

	float xi=xrnd(s);
	if(e.fldr<0) xi*=2*FPI;
	else{
	  int r=__float2int_rd(e.fldr/360)+1;
	  int s=__float2int_rd(xi*r);
	  xi=(e.fldr+s*360/r)*fcv;
	}
	sincosf(xi, &n.y, &n.x);

	if(e.ka>0){
	  float FLZ, FLR;
	  sincosf(fcv*30.f, &FLZ, &FLR);
	  FLZ*=OMR, FLR*=OMR;
	  r.x+=FLR*n.x;
	  r.y+=FLR*n.y;
	  r.z+=FLZ;
	  r.w+=OMR*w->ocm;
	}

	sincosf(e.up, &n.z, &xi);
	n.x*=xi; n.y*=xi;

	if(e.ka>0){
	  do{ xi=1+e.ka*logf(xrnd(s)); } while (xi<-1);
	  float si=sqrtf(1-xi*xi); rotate(xi, si, n, s);
	}
      }
      else{
	photon p=e.pz[i/OVER]; r=p.r; n=p.n;
	float & l=p.l;
#ifndef TALL
	niw=p.q;
#endif

	if(l<0){
	  float xi;
	  if(e.ka>0){
	    do{ xi=1+e.ka*logf(xrnd(s)); } while (xi<-1);
	    float si=sqrtf(1-xi*xi); rotate(xi, si, n, s);
	  }
	}
	else{
	  if(l>0) l*=xrnd(s);
#ifdef LONG
	  else if(p.b>0) l=p.b*mrnd(p.a, s);
#endif
	  if(l>0){
	    r.w+=e.ocv*l;
	    r.x+=n.x*l; r.y+=n.y*l; r.z+=n.z*l;
	  }

#ifdef ANGW
	  if(p.f<xrnd(s)){
	    const float a=0.39f, b=2.61f;
	    const float I=1-expf(-b*exp2f(a));
	    float cs=max(1-powf(-logf(1-xrnd(s)*I)/b, 1/a), -1.0f);
	    float si=sqrtf(1-cs*cs); rotate(cs, si, n, s);
	  }
#endif
	  rotate(w->coschr, w->sinchr, n, s);
	}
      }

#ifdef TALL
      pbuf f; f.r=r, f.n=n, f.q=j; e.bf[i]=f;
  }
#ifndef XCPU
  __threadfence_block();
#endif

  for(unsigned int i=idx; i<num; TOT==0 && (XINC)){
    int om=-1;
    if(TOT==0){ // initialize photon
      pbuf f=e.bf[i];
      r=f.r; n=f.n; w=&e.z->w[f.q];
      if(e.type<=0) niw=e.pz[i/OVER].q;
#endif
      om=e.fla;

      TOT=-logf(xrnd(s)), IFH(SCA,sca)=0;
    }

#ifdef HOLE
    if(SCA==0) SCA=-logf(xrnd(s)), old=om;
    float sca, tot;
#else
    if(sca==0){ // get distance for overburden
      float SCA=-logf(xrnd(s)); old=om;
      float tot;
#endif
      float z = r.z;
#ifdef TILT
      z -= zshift(e, r);
#endif

      float nr=1.f;
#ifdef ANIZ
      if(e.k>0){
	float n1= e.azx*n.x+e.azy*n.y;
	float n2=-e.azy*n.x+e.azx*n.y;
	float n3= n.z;

	float s1=n1*n1, l1=e.k1*e.k1;
	float s2=n2*n2, l2=e.k2*e.k2;
	float s3=n3*n3, l3=e.kz*e.kz;

	float B2=nr/l1+nr/l2+nr/l3;
	float nB=s1/l1+s2/l2+s3/l3;
	float An=s1*l1+s2*l2+s3*l3;

	nr=(B2-nB)*An/2;
	TOT/=nr;
      }
#endif

#ifdef HOLE
    {
#endif
      int i=__float2int_rn((z-e.hmin)*e.rdh);
      if(i<0) i=0; else if(i>=e.size) i=e.size-1;
      float h=e.hmin+i*e.dh; // middle of the layer
      h=n.z<0?h-e.hdh:h+e.hdh;

      float ais=(n.z*SCA-(h-z)*w->z[i].sca)*e.rdh;
      float aia=(n.z*TOT-(h-z)*w->z[i].abs)*e.rdh;

      int j=i;
      if(n.z<0) for(; j>0 && ais<0 && aia<0; h-=e.dh, ais+=w->z[j].sca, aia+=w->z[j].abs) --j;
      else for(; j<e.size-1 && ais>0 && aia>0; h+=e.dh, ais-=w->z[j].sca, aia-=w->z[j].abs) ++j;

      if(i==j || fabsf(n.z)<XXX) sca=SCA/w->z[j].sca, tot=TOT/w->z[j].abs;
      else sca=(ais*e.dh/w->z[j].sca+h-z)/n.z, tot=(aia*e.dh/w->z[j].abs+h-z)/n.z;

      // get overburden for distance
      if(tot<sca) sca=tot, IFH(tot,TOT)=0; else IFH(tot=,TOT=nr*)(tot-sca)*w->z[j].abs;
    }

    om=-1;
    float del=sca;
#ifdef HOLE
    float hi=sca, hf=0;
#endif
    { // sphere
#ifndef HOLE
      float & sca = del;
#endif
      float2 ri, rf, pi, pf;

      ri.x=r.x; rf.x=r.x+sca*n.x;
      ri.y=r.y; rf.y=r.y+sca*n.y;

      ctr(e, ri, pi); ctr(e, rf, pf);

      ri.x=min(pi.x, pf.x)-e.rx; rf.x=max(pi.x, pf.x)+e.rx;
      ri.y=min(pi.y, pf.y)-e.rx; rf.y=max(pi.y, pf.y)+e.rx;

      int2 xl, xh;

      xl.x=min(max(__float2int_rn((ri.x-e.cl[0])*e.crst[0]), 0), e.cn[0]);
      xh.x=max(min(__float2int_rn((rf.x-e.cl[0])*e.crst[0]), e.cn[0]-1), -1);

      xl.y=min(max(__float2int_rn((ri.y-e.cl[1])*e.crst[1]), 0), e.cn[1]);
      xh.y=max(min(__float2int_rn((rf.y-e.cl[1])*e.crst[1]), e.cn[1]-1), -1);

      for(int i=xl.x, j=xl.y; i<=xh.x && j<=xh.y; ++j<=xh.y?:(j=xl.y,i++)) for(unsigned char k=e.is[i][j]; k!=0x80; ){
	unsigned char m=e.ls[k];
	line & s = e.sc[m&0x7f];
	k=m&0x80?0x80:k+1;

	float b=0, c=0, dr;
	dr=s.x-r.x;
	b+=n.x*dr; c+=dr*dr;
	dr=s.y-r.y;
	b+=n.y*dr; c+=dr*dr;

	float np=1-n.z*n.z;
	float D=b*b-(c-s.r*s.r)*np;
	if(D>=0){
	  D=sqrtf(D);
	  float h1=b-D, h2=b+D;
	  if(h2>=0 && h1<=sca*np){
	    if(np>XXX){
	      h1/=np, h2/=np;
	      if(h1<0) h1=0; if(h2>sca) h2=sca;
	    }
	    else h1=0, h2=sca;
	    h1=r.z+n.z*h1, h2=r.z+n.z*h2;
	    float zl, zh;
	    if(n.z>0) zl=h1, zh=h2;
	    else zl=h2, zh=h1;

	    int omin=0, omax=s.max;
	    int n1=s.n-omin+min(omax+1, max(omin, __float2int_ru(omin-(zh-s.dl-s.h)*s.d)));
	    int n2=s.n-omin+max(omin-1, min(omax, __float2int_rd(omin-(zl-s.dh-s.h)*s.d)));

	    for(int l=n1; l<=n2; l++) if(l!=old){
#ifdef OFLA
	      if(l==e.fla) continue;
#endif
	      const DOM & dom=oms[l];
	      float b=0, c=0, dr;
	      dr=dom.r[0]-r.x;
	      b+=n.x*dr; c+=dr*dr;
	      dr=dom.r[1]-r.y;
	      b+=n.y*dr; c+=dr*dr;
	      dr=dom.r[2]-r.z;
	      b+=n.z*dr; c+=dr*dr;
	      float D=b*b-c+e.R2;
	      if(D>=0){
		float h=b-sqrtf(D)*e.zR;
		if(h>0 && h<=del) om=l, del=h;
	      }
	    }
	  }
#ifdef HOLE
	  if(e.hr>0){
	    float D=b*b-(c-e.hr2)*np;
	    if(D>0){
	      D=sqrtf(D);
	      float h1=b-D, h2=b+D;
	      if(h2>=0 && h1<=sca*np){
		if(np>XXX){
		  h1/=np, h2/=np;
		  if(h1<0) h1=0; if(h2>sca) h2=sca;
		}
		else h1=0, h2=sca;
		if(h1<hi && h2>sqrtf(XXX)*e.hr) hi=h1, hf=h2;
	      }
	    }
	  }
#endif
	}
      }
    }

#ifdef HOLE
    float fin=min(del, hi);
    bool hole=fin<sca;
    if(hole){
      { // get overburden for distance
	float xs=0, xa=0;
	int i=__float2int_rn((z-e.hmin)*e.rdh);
	if(i<0) i=0; else if(i>=e.size) i=e.size-1;

	float y = z + n.z*fin;
	int j=__float2int_rn((y-e.hmin)*e.rdh);
	if(j<0) j=0; else if(j>=e.size) j=e.size-1;

	if(i==j || fabsf(n.z)<XXX) xs=fin*w->z[i].sca, xa=fin*w->z[i].abs;
	else{
	  int k=j;
	  float h=e.hmin+i*e.dh, g=e.hmin+j*e.dh;
	  if(n.z<0){
	    h-=e.hdh, g+=e.hdh;
	    while(++k<i) xs-=w->z[k].sca, xa-=w->z[k].abs;
	  }
	  else{
	    h+=e.hdh, g-=e.hdh;
	    while(--k>i) xs+=w->z[k].sca, xa+=w->z[k].abs;
	  }
	  xs=((y-g)*w->z[j].sca+(h-z)*w->z[i].sca+e.dh*xs)/n.z;
	  xa=((y-g)*w->z[j].abs+(h-z)*w->z[i].abs+e.dh*xa)/n.z;
	}
	SCA-=xs, TOT-=xa;
      }
      TOT*=nr;

      if(hi<del){
	fin=min(hi+min(SCA/e.hs, TOT/e.ha), hf);
	if(fin<del) del=fin, om=-1;
	fin-=hi; SCA-=fin*e.hs, TOT-=fin*e.ha;
      }
    }
    else SCA=0, TOT=tot*nr;
#else
    sca-=del;
#endif

    { // advance
      r.x+=del*n.x;
      r.y+=del*n.y;
      r.z+=del*n.z;
      r.w+=del*w->ocm;
    }

#ifndef XCPU
    if(!isfinite(TOT) || !isfinite(IFH(SCA,sca))) ed->bmp[atomicAdd(&ed->ab, 1)%4]=smid(), TOT=0, om=-1;
#endif

    float xi=xrnd(s);
    if(om!=-1){
      bool flag=true;
      hit h; h.i=om; h.t=r.w; h.n=niw; h.z=w->wvl;

#ifdef ASENS
      float sum;
      {
	float & x = n.z;
	float y=1;
	sum=e.s[0];
	for(int i=1; i<ANUM; i++){ y*=x; sum+=e.s[i]*y; }
      }

      flag=e.mas*xi<sum;
#endif
      if(e.type>0){
	float dt=0, dr;
	const DOM & dom=oms[om];
	for(int i=0; i<3; i++, dt+=dr*dr) dr=dom.r[i]-e.r[i];
	if(h.t<(sqrtf(dt)-OMR)*w->ocm) flag=false;
      }

      if(flag){
	unsigned int j = atomicAdd(&ed->hidx, 1);
	if(j<e.hnum) e.hits[j]=h;
      }

      if(e.zR==1) TOT=0; else old=om;
    }
    else if(TOT<XXX) TOT=0;
#ifdef HOLE
    else if(SCA<XXX){
      SCA=0;
      float sf, g, g2, gr;
      if(hole){
	sf=e.SF, g=e.G, g2=e.G2, gr=e.GR;
      }
      else{
	sf=e.sf, g=e.g, g2=e.g2, gr=e.gr;
      }
#else
    else{
      float &sf=e.sf, &g=e.g, &g2=e.g2, &gr=e.gr;
#endif

      if(xi>sf){
	xi=(1-xi)/(1-sf);
	xi=2*xi-1;
	if(g!=0){
	  float ga=(1-g2)/(1+g*xi);
	  xi=(1+g2-ga*ga)/(2*g);
	}
      }
      else{
	xi/=sf;
	xi=2*powf(xi, gr)-1;
      }

      if(xi>1) xi=1; else if(xi<-1) xi=-1;

#ifdef ANIZ
      if(e.k>0 IFH(&& !hole,)){
	float n1=( e.azx*n.x+e.azy*n.y)*e.k1;
	float n2=(-e.azy*n.x+e.azx*n.y)*e.k2;
	float nx=n1*e.azx-n2*e.azy;
	float ny=n1*e.azy+n2*e.azx;
	float nz=n.z*e.kz;
	float r=rsqrtf(nx*nx+ny*ny+nz*nz);
	n.x=r*nx, n.y=r*ny, n.z=r*nz;
      }
#endif

      float si=sqrtf(1-xi*xi);
      rotate(xi, si, n, s);

#ifdef ANIZ
      if(e.k>0 IFH(&& !hole,)){
	float n1=( e.azx*n.x+e.azy*n.y)/e.k1;
	float n2=(-e.azy*n.x+e.azx*n.y)/e.k2;
	float nx=n1*e.azx-n2*e.azy;
	float ny=n1*e.azy+n2*e.azx;
	float nz=n.z/e.kz;
	float r=rsqrtf(nx*nx+ny*ny+nz*nz);
	n.x=r*nx, n.y=r*ny, n.z=r*nz;
      }
#endif
    }
  }

  {
    e.z->rs[s.w]=s.x | (unsigned long long) s.y << 32;
#ifndef XCPU
    __syncthreads();
    if(threadIdx.x==0){
      e.tx=clock();
      atomicMin(&ed->tn, e.tx-e.tn);
      atomicMax(&ed->tx, e.tx-e.tn);
    }
    __threadfence();
#endif
  }

}
